#include <algorithm>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

__global__ void kernel_function_a(const double *A, const double *x, double *y, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        double temp = 0.0;
        for (int j = 0; j < N; j++) {
            temp += A[idx * N + j] * x[idx]; 
        }
        y[idx] = temp;
    }
}

__global__ void kernel_function_b(const double a, const double *u, const double *v, double *x, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        x[idx] = a * u[idx] + v[idx];
    }
}

__global__ void kernel_function_c(const double s, const double *x, const double *y, double *z, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        if (idx % 2 == 0) {
            z[idx] = s * x[idx] + y[idx]; 
        } else {
            z[idx] = x[idx] + y[idx];   
        }
    }
}

__global__ void kernel_function_d(const double *u, const double *v, double *result, int N) {
    extern __shared__ double sdata[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load data into shared memory
    double sum = 0;
    if (idx < N) {
        sum = u[idx] * v[idx];
    }
    sdata[tid] = sum;

    __syncthreads();

    // Reduction within a block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Sum from each block is written back to result
    if (tid == 0) {
        atomicAdd(result, sdata[0]);
    }
}

void init_datastructures(double *u, double *v, double *A, const int N) {
    for (unsigned int i = 0; i < N; i++) {
        u[i] = static_cast<double>(i%2);
        v[i] = static_cast<double>(i%4);
    }

    for (unsigned int i = 0; i < N * N; i++) {
        A[i] = static_cast<double>(i%8);
    }
}

void print_results_to_file(const double s, const double *x, const double *y,
                           const double *z, const double *A, const long long n,
                           std::ofstream &File) {

    unsigned int N = std::min(n, static_cast<long long>(30));

    File << "N: "
        << "\n"
        << n << "\n";

    File << "s: "
        << std::fixed
        << std::setprecision(1)
        << "\n"
        << s << "\n";

    File << "x: "
        << "\n";
    for (unsigned int i = 0; i < N; i++) {
        File << x[i] << " ";
    }
    File << "\n";

    File << "y: "
        << "\n";
    for (unsigned int i = 0; i < N; i++) {
        File << y[i] << " ";
    }
    File << "\n";

    File << "z: "
        << "\n";
    for (unsigned int i = 0; i < N; i++) {
        File << z[i] << " ";
    }
    File << "\n";
}

int main(int argc, char **argv) {
    long long N;

    if (argc == 2) {
        N = std::stoi(argv[1]);
    } else {
        std::cout << "Error: Missing problem size N. Please provide N as "
                    "commandline parameter. Usage example for N=10: "
                    "./number_crunching 10"
                << std::endl;
        exit(0);
    }

    double *u = new double[N];
    double *v = new double[N];
    double *A = new double[N * N];
    double *x = new double[N];
    double *y = new double[N];
    double *z = new double[N];
    double *d_u, *d_v, *d_A, *d_x, *d_y, *d_z, *d_result;
    double s;

    // Allocate memory on the device
    hipMalloc(&d_u, N * sizeof(double));
    hipMalloc(&d_v, N * sizeof(double));
    hipMalloc(&d_A, N * N * sizeof(double));
    hipMalloc(&d_x, N * sizeof(double));
    hipMalloc(&d_y, N * sizeof(double));
    hipMalloc(&d_z, N * sizeof(double));
    hipMalloc(&d_result, sizeof(double));

    // Initialize data (host)
    init_datastructures(u, v, A, N);

    // Copy data to the device
    hipMemcpy(d_u, u, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);

    // Execute kernels
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    kernel_function_b<<<numBlocks, blockSize>>>(2.0, d_u, d_v, d_x, N);
    kernel_function_a<<<numBlocks, blockSize>>>(d_A, d_x, d_y, N);
    double zero = 0.0; 
    hipMemcpy(d_result, &zero, sizeof(double), hipMemcpyHostToDevice);
    kernel_function_d<<<numBlocks, blockSize, blockSize * sizeof(double)>>>(d_u, d_v, d_result, N);
    hipMemcpy(&s, d_result, sizeof(double), hipMemcpyDeviceToHost); 
    kernel_function_c<<<numBlocks, blockSize>>>(s, d_x, d_y, d_z, N);

    hipDeviceSynchronize();

    // Copy results back to host
    hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(z, d_z, N * sizeof(double), hipMemcpyDeviceToHost);

    std::ofstream File("partial_results.out");
    print_results_to_file(s, x, y, z, A, N, File);

    std::cout << "For correctness checking, partial results have been written to "
                "partial_results.out"
                << std::endl;

    delete[] u; 
    delete[] v; 
    delete[] A; 
    delete[] x; 
    delete[] y; 
    delete[] z;
    hipFree(d_u); 
    hipFree(d_v); 
    hipFree(d_A); 
    hipFree(d_x); 
    hipFree(d_y); 
    hipFree(d_z); 
    hipFree(d_result);

    return EXIT_SUCCESS;
}