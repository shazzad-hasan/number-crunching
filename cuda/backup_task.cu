#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <iomanip>

// Handle CUDA errors
void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        exit(-1);
    }
}

// Kernel to replicate function_a behavior (matrix-vector multiplication)
__global__ void kernel_function_a(const double *A, const double *x, double *y, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        double temp = 0.0;
        for (int j = 0; j < N; j++) {
            temp += A[idx * N + j] * x[idx];  // Corrected to use x[j] for proper matrix-vector multiplication
        }
        y[idx] = temp;
    }
}

// Kernel to replicate function_b behavior (vector scaling and addition)
__global__ void kernel_function_b(const double a, const double *u, const double *v, double *x, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        x[idx] = a * u[idx] + v[idx];
    }
}

// Kernel to replicate function_c behavior (conditional vector operations)
__global__ void kernel_function_c(const double s, const double *x, const double *y, double *z, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        z[idx] = (idx % 2 == 0) ? s * x[idx] + y[idx] : x[idx] + y[idx];
    }
}

__global__ void kernel_function_d(const double *u, const double *v, double *result, int N) {
    extern __shared__ double sdata[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load data into shared memory and compute partial sum
    double sum = (idx < N) ? u[idx] * v[idx] : 0;
    sdata[tid] = sum;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the result for this block to global mem
    if (tid == 0) {
        atomicAdd(result, sdata[0]);
    }
}

void init_datastructures(double *u, double *v, double *A, const int N) {
    for (unsigned int i = 0; i < N; i++) {
        u[i] = static_cast<double>(i % 2);
        v[i] = static_cast<double>(i % 4);
    }
    for (unsigned int i = 0; i < N * N; i++) {
        A[i] = static_cast<double>(i % 8);
    }
}

void print_results_to_file(const double s, const double *x, const double *y, const double *z, const double *A, const long long n, std::ofstream &File) {
    unsigned int N = std::min(n, static_cast<long long>(30));
    File << "N: \n" << n << "\n";
    File << "s: " << std::fixed << std::setprecision(1) << "\n" << s << "\n";
    File << "x: \n";
    for (unsigned int i = 0; i < N; i++) {
        File << x[i] << " ";
    }
    File << "\n";
    File << "y: \n";
    for (unsigned int i = 0; i < N; i++) {
        File << y[i] << " ";
    }
    File << "\n";
    File << "z: \n";
    for (unsigned int i = 0; i < N; i++) {
        File << z[i] << " ";
    }
    File << "\n";
}

// Include your kernel definitions and any utility functions here

int main(int argc, char **argv) {
    if (argc != 2) {
        std::cerr << "Error: Missing problem size N. Please provide N as a command-line parameter. Usage example for N=10: ./number_crunching_task 10" << std::endl;
        return EXIT_FAILURE;
    }

    const int N = std::stoi(argv[1]);
    const int blockSize = 256; // Define the optimal block size based on your GPU architecture

    double *u = new double[N];
    double *v = new double[N];
    double *A = new double[N * N];
    double *x = new double[N];
    double *y = new double[N];
    double *z = new double[N];
    double *d_u, *d_v, *d_A, *d_x, *d_y, *d_z, *d_result;
    double s = 0.0;

    checkCuda(hipMalloc(&d_u, N * sizeof(double)));
    checkCuda(hipMalloc(&d_v, N * sizeof(double)));
    checkCuda(hipMalloc(&d_A, N * N * sizeof(double)));
    checkCuda(hipMalloc(&d_x, N * sizeof(double)));
    checkCuda(hipMalloc(&d_y, N * sizeof(double)));
    checkCuda(hipMalloc(&d_z, N * sizeof(double)));
    checkCuda(hipMalloc(&d_result, sizeof(double)));

    // Initialize data on host and copy to device
    init_datastructures(u, v, A, N);
    checkCuda(hipMemcpy(d_u, u, N * sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_v, v, N * sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_result, &s, sizeof(double), hipMemcpyHostToDevice));  // Initialize result on device

    // Prepare streams
    hipStream_t stream1, stream2, stream3, stream4;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);

    // Launch kernels in their respective streams
    kernel_function_b<<<(N + blockSize - 1) / blockSize, blockSize, 0, stream1>>>(2.0, d_u, d_v, d_x, N);
    kernel_function_a<<<(N + blockSize - 1) / blockSize, blockSize, 0, stream2>>>(d_A, d_x, d_y, N);
    int numBlocks = min((N + blockSize - 1) / blockSize, 1024);
    kernel_function_d<<<numBlocks, blockSize, blockSize * sizeof(double), stream3>>>(d_u, d_v, d_result, N);
    hipStreamSynchronize(stream3);  // Ensure reduction is complete before proceeding

    // Additional reduction step if numBlocks > 1
    if (numBlocks > 1) {
        // Call another reduction kernel to sum results from each block, store final result in `d_result`
    }
    hipMemcpyAsync(&s, d_result, sizeof(double), hipMemcpyDeviceToHost, stream3);
    hipStreamSynchronize(stream3);  // Ensure 's' is updated on host

    kernel_function_c<<<(N + blockSize - 1) / blockSize, blockSize, 0, stream4>>>(s, d_x, d_y, d_z, N);
    hipStreamSynchronize(stream4);  // Wait for all operations to complete

    // Copy results back to host
    hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(z, d_z, N * sizeof(double), hipMemcpyDeviceToHost);

    // Output results to file
    std::ofstream File("partial_results.out");
    print_results_to_file(s, x, y, z, A, N, File);
    File.close();

    std::cout << "For correctness checking, partial results have been written to partial_results.out" << std::endl;

    // Cleanup
    delete[] u; delete[] v; delete[] A; delete[] x; delete[] y; delete[] z;
    hipFree(d_u); hipFree(d_v); hipFree(d_A); hipFree(d_x); hipFree(d_y); hipFree(d_z); hipFree(d_result);
    hipStreamDestroy(stream1); hipStreamDestroy(stream2); hipStreamDestroy(stream3); hipStreamDestroy(stream4);

    return EXIT_SUCCESS;
}
